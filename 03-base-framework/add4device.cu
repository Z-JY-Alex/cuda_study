
#include <hip/hip_runtime.h>
#include <math.h>
#include <stdio.h>

const double EPSILON = 1.0e-15;
const double a = 1.23;
const double b = 2.34;
const double c = 3.57;
__global__ void add1(const double *x, const double *y, double *z, const int N);
__global__ void add2(const double *x, const double *y, double *z, const int N);
__global__ void add3(const double *x, const double *y, double *z, const int N);

void check(const double *z, const int N);

int main(void)
{
    const int N = 100000001;
    const int M = sizeof(double) * N;
    double *h_x = (double*) malloc(M);
    double *h_y = (double*) malloc(M);
    double *h_z = (double*) malloc(M);

    for (int n = 0; n < N; ++n)
    {
        h_x[n] = a;
        h_y[n] = b;
    }

    double *d_x, *d_y, *d_z;
    hipMalloc((void **)&d_x, M);
    hipMalloc((void **)&d_y, M);
    hipMalloc((void **)&d_z, M);
    hipMemcpy(d_x, h_x, M, hipMemcpyHostToDevice);
    hipMemcpy(d_y, h_y, M, hipMemcpyHostToDevice);

    const int block_size = 128;
    const int grid_size = (N-1) /block_size +1;
    add1<<<grid_size, block_size>>>(d_x, d_y, d_z, N);
    hipMemcpy(h_z, d_z, M, hipMemcpyDeviceToHost);
    check(h_z, N);

    add2<<<grid_size, block_size>>>(d_x, d_y, d_z, N);
    hipMemcpy(h_z, d_z, M, hipMemcpyDeviceToHost);
    check(h_z, N);

    add3<<<grid_size, block_size>>>(d_x, d_y, d_z, N);
    hipMemcpy(h_z, d_z, M, hipMemcpyDeviceToHost);
    check(h_z, N);

    free(h_x);
    free(h_y);
    free(h_z);
    hipFree(d_x);
    hipFree(d_y);
    hipFree(d_z);
    return 0;
}

__device__ double add1_device(const double x, const double y)
{
    return (x+y);
}


__global__ void add1(const double *x, const double *y, double *z, const int N)
{
    const int n = blockDim.x * blockIdx.x + threadIdx.x;
    if (n < N)
    {
        z[n] = add1_device(x[n], y[n]);
    }
}

__device__ double add2_device(const double x, const double y, double *z)
{
    *z = x+y;
}

__global__ void add2(const double *x, const double *y, double *z, const int N)
{
    const int n = blockDim.x * blockIdx.x + threadIdx.x; 
    if (n < N)
    {
        add2_device(x[n], y[n], &z[n]);
    }
}

__device__ double add3_device(const double x, const double y, double &z)
{
    z = x+y;
}

__global__ void add3(const double *x, const double *y, double *z, const int N)
{
    const int n = blockDim.x * blockIdx.x + threadIdx.x;
    if (n < N)
    {
        add3_device(x[n], y[n], z[n]);
    }
}

void check(const double *z, const int N)
{
    bool has_error = false;
    for (int n = 0; n < N; ++n)
    {
        if (fabs(z[n] - c) > EPSILON)
        {
            has_error = true;
        }
    }
    printf("%s\n", has_error ? "Has errors" : "No errors");
}