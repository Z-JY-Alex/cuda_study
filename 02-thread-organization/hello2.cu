
#include <hip/hip_runtime.h>
#include <stdio.h>
// #include <iostream>

__global__ void hello_from_gpu()
{
    printf("Hello World from the GPU!\n");
    // std::cout << "Hello World from the GPU\n";
}

int main(){
    hello_from_gpu<<<1, 1>>>();
    hipDeviceSynchronize();
    return 0;
}